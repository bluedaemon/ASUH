#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"



__global__ void cudamin(int *arrayD,int *numberD) {
extern __shared__ int sdata[];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
sdata[tid] = arrayD[i];
__syncthreads();

for(unsigned int s=1; s < blockDim.x; s *= 2) {
if (tid % (2*s) == 0) {
sdata[tid]= (sdata[tid]<sdata[tid+s]) ? sdata[tid]: sdata[tid+s];
}
__syncthreads();
}

if (tid == 0) numberD[blockIdx.x] = sdata[0];

}

void CUDAmin(int* argv, int* numberH){
	const int sz=100*sizeof(int);
	int *numberD;
	int *arrayD;
	int i=0;
	
	/*char line[80];
	FILE *a;
	a=fopen(argv,"r");
	while((fgets(line, 80, a) != NULL) && i<100){
		sscanf (line, "%d", &numberH[i]);
		array[i]=numberH[i];
		i++;
  	}*/

	hipMalloc((void **)&arrayD,sz);
	hipMalloc((void **)&numberD,sz);
	hipMemcpy(arrayD,argv,sz,hipMemcpyHostToDevice);
	dim3 gridDim(5);
	dim3 blockDim(20);	
	cudamin<<<gridDim,blockDim>>>(arrayD,numberD);

	hipDeviceSynchronize();
	hipMemcpy(numberH,numberD,sz,hipMemcpyDeviceToHost);
	hipFree(arrayD);
	hipFree(numberD);
}