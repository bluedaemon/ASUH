#include "hip/hip_runtime.h"
/*
This function takes the number of "vectors" and number of "places" and a two file names as arguments
example : ./max 10 10 input.csv output.cs
*/

#include <hip/hip_runtime.h>
#include "fileio.h"

__global__ void cmax(int *arrayD,int* answerD,int size) {
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i<size){
	unsigned int tid = threadIdx.x;
	extern __shared__ int sdata[];
	
	sdata[tid] = arrayD[i];
	__syncthreads();
	unsigned int s;
	for (s=(blockDim.x%2==0 ? blockDim.x>>1 : (blockDim.x>>1)+1); s>0; s=(s%2==0 ? s>>1 : (s>>1)+1)) {
		if(tid < s){
			sdata[tid]= sdata[tid]>sdata[tid+s] ? sdata[tid]: sdata[tid+s];
		}
	__syncthreads();
	if(s==1)break;
	}
	if (tid == 0) answerD[blockIdx.x] = sdata[0];
}
}

int main(int argc, char** argv){
	hipError_t cudaStatus;
	const unsigned int vector=atoi(argv[1]);
	const unsigned int place=atoi(argv[2]);
	const unsigned int sz=vector*place*sizeof(int);
	int *answerD,*answerH,*arrayD,*inputarray,i;
	int **answer=createArray(1,1);
	const int blocka=place*vector%512==0 ? place*vector/512 : (place*vector/512)+1;

	inputarray=(int*)malloc(sz);
	if(inputarray==NULL){
		printf("failed malloc\n");
		exit(-1);
	}

	answerH=(int*)malloc(sizeof(int)*blocka);
	if(answerH==NULL){
		printf("failed malloc\n");
		exit(-1);
	}

	readFileCUDA(vector,place,inputarray,argv[3]);

	cudaStatus=hipDeviceSetLimit(hipLimitMallocHeapSize,1073741824);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipDeviceSetLimit failed!,%s\n",hipGetErrorString(cudaStatus));
		exit(-1);
	}

	cudaStatus=hipMalloc((void **)&arrayD,sz);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc failed!,%s\n",hipGetErrorString(cudaStatus));
		exit(-1);
 	}

	cudaStatus=hipMalloc(&answerD,sizeof(int)*blocka);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMalloc failed!,%s\n",hipGetErrorString(cudaStatus));
		exit(-1);
	}

	cudaStatus=hipMemcpy(arrayD,inputarray,sz,hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess){
		fprintf(stderr, "hipMemcpy failed!,%s\n",hipGetErrorString(cudaStatus));
		exit(-1);
	}

	dim3 blockDim(512);
	dim3 gridDim(blocka);

	cmax<<<gridDim,blockDim,512*sizeof(int)>>>(arrayD,answerD,vector*place);

	if(hipSuccess != (cudaStatus=hipGetLastError())){
		printf("Kernel error! %s\n" ,hipGetErrorString(cudaStatus));
		exit(-1);
	}


  	cudaStatus=hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "devicesync failed!,%s\n",hipGetErrorString(cudaStatus));
		exit(-1);
	}
	
	cudaStatus=hipMemcpy(answerH,answerD,sizeof(int)*blocka,hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "memcpy failed!,%s\n",hipGetErrorString(cudaStatus));
		exit(-1);
	}

	answer[0][0]=answerH[0];
	if(blocka!=1){
		for(i=1;i<(blocka);i++){
			answer[0][0]=(answerH[i]>answer[0][0] ? answerH[i] : answer[0][0]);
		}
	}
	writeFile(1,1,answer,argv[4]);
	freeArray(1,1,answer);
	free(answerH);
	free(inputarray);
	hipFree(arrayD);
	return 0;
}
