#include "hip/hip_runtime.h"
/*
This function takes the number of "vectors" and number of "places" and a two file names as arguments
example : ./max 10 10 input.csv output.cs
*/

#include <hip/hip_runtime.h>
#include "fileio.h"

__global__ void cmax(int *arrayD,int* answerD,int size) {
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
if(i<size){

unsigned int tid = threadIdx.x;
extern __shared__ int sdata[];

sdata[tid] = arrayD[i];
__syncthreads();

for (unsigned int s=(size%2==0 ? size>>1 : size>>1+1); s>0; s=(s%2==0 ? s>>1 : (s>>1)+1)) {
	if(tid < s){printf("%d>%d\n",sdata[tid],sdata[tid+s]);
		sdata[tid]= sdata[tid]>sdata[tid+s] ? sdata[tid]: sdata[tid+s];
	}
__syncthreads();
if(s==1)break;
}

if (tid == 0) answerD[blockIdx.x] = sdata[0];
}
}

int main(int argc, char** argv){
	const int vector=atoi(argv[1]);
	const int place=atoi(argv[2]);
	const int sz=vector*place*sizeof(int);
	int *answerD,*answerH,*arrayD,*inputarray,i;
	int **answer=createArray(1,1);
	const int blocka=place*vector%256==0 ? place*vector/256 : (place*vector/256)+1;

	inputarray=(int*)malloc(sz);
	answerD=(int*)malloc(sizeof(int)*blocka);
	answerH=(int*)malloc(sizeof(int)*blocka);
	readFileCUDA(vector,place,inputarray,argv[3]);

	hipMalloc((void **)&arrayD,sz);
	hipMalloc(&answerD,sizeof(int)*blocka);
	hipMemcpy(arrayD,inputarray,sz,hipMemcpyHostToDevice);

	dim3 blockDim(256);
	dim3 gridDim(blocka);

	cmax<<<gridDim,blockDim,sz>>>(arrayD,answerD,vector*place);
	
	hipDeviceSynchronize();
	hipMemcpy(answerH,answerD,sizeof(int)*blocka,hipMemcpyDeviceToHost);

	if(blocka==1){
		answer[0][0]=answerH[0];
	}

	else{
	for(i=0;i<blocka-1;i++){
		answer[0][0]=(answerH[i]>answerH[i+1] ? answerH[i] : answerH[i+1]);
	}
	}
	writeFile(1,1,answer,argv[4]);
	free(*answer);
	free(answer);
	free(inputarray);
	hipFree(arrayD);
	return 0;
}
