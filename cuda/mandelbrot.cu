#include "hip/hip_runtime.h"
/*
	This is the mandelbrot function
	It takes the number of "vectors" and number of "places" and one file name as arguments
	The resulting output file will contain a mandelbrot fractal
	example: ./mandelbrot 100 100 out.csv
*/
#include "fileio.h"
#include <hip/hip_runtime.h>
#include <math.h>

typedef struct complextype{
        float real, imag;
} Compl;

__global__ void calc(int*,int,int);

int main (int argc,char ** argv)
{
	
        int i, j;
	const int xdim=atoi(argv[1]);
	const int ydim=atoi(argv[2]);
	
	dim3 blockDim(16,16);
	const int gridx= xdim%blockDim.x==0 ? xdim/blockDim.x : xdim/blockDim.x+1; 
	const int gridy= ydim%blockDim.y==0 ? ydim/blockDim.y : ydim/blockDim.y+1; 
	dim3 gridDim(gridx,gridy);
	int* pixels;
	int* pixels2=(int*) malloc(sizeof(int)*xdim*ydim);
	int **grid = createArray(xdim,ydim);

	hipMalloc((void**)&pixels,xdim*ydim*sizeof(int));

	calc<<<gridDim, blockDim>>>(pixels,xdim,ydim);

	hipDeviceSynchronize();

	hipMemcpy(pixels2,pixels,xdim*ydim*sizeof(int),hipMemcpyDeviceToHost);

	for(i=0;i<xdim;i++){
	for(j=0;j<ydim;j++){
		grid[i][j]=pixels2[i*ydim+j];
	}
	}
	writeFile(xdim, ydim, grid, argv[3]);		

	hipFree(pixels);

}

__global__ void calc(int* pixels, int xdim, int ydim){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y*blockDim.y+threadIdx.y;
	if( i<xdim*ydim && j<xdim*ydim){
		Compl	z, c;
		int k;
		float	lengthsq, temp;
	
        	z.real = z.imag = 0.0;
        	c.real = ((float) j - (ydim/2.0))/(ydim/4.0);  
		c.imag = ((float) i - (xdim/2.0))/(xdim/4.0);
        	k = 0;
	
		do{                                           
        	 	temp = z.real*z.real - z.imag*z.imag + c.real;
        	 	z.imag = 2.0*z.real*z.imag + c.imag;
        	 	z.real = temp;
        	 	lengthsq = z.real*z.real+z.imag*z.imag;
        	 	k++;
        	} while (lengthsq < 4.0 && k < 100);
	
		pixels[i*ydim+j]=(k==100);
}
}	

